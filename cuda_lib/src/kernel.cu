#include "hip/hip_runtime.h"
#include "kernel.h"

using namespace cv::cuda;


inline __device__ __host__ int iDivUp( int a, int b ) { return (a % b != 0) ? (a / b + 1) : (a / b);}

__global__ void image2Matrix_kernel(int width, int height,  PtrStepSz<uchar3> image, float* matrix)
{


	const int w = blockIdx.x * blockDim.x + threadIdx.x;
	const int h = blockIdx.y * blockDim.y + threadIdx.y;
    
    
    if (w < width && h < height)
    {
        uchar3 v = image(h,w);
        *(matrix + 0*height*width + h*width + w) = float(v.x);
        *(matrix + 1*height*width + h*width + w) = float(v.y);
        *(matrix + 2*height*width + h*width + w) = float(v.z);
    }

}
//, hipStream_t &stream
void gpu_image2Matrix(int width, int height,  cv::cuda::GpuMat & image, float* matrix)
{
     /*
        image : input image in GpuMat format, WHC arrangement and BGR order
        matrix: gpu float array, CHW and RGB order
    */
    //dim3 block(width, height); // width * height blocks, 1 thread each
    const dim3 blockDim(32, 32);
    const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));
    image2Matrix_kernel<<<gridDim,blockDim>>>(width,height,image,matrix);
}

